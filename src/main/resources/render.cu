
#include <hip/hip_runtime.h>

// 0 - x;
// 1 - y;
// 2 - z;
// 3 - mass;
// 4 - radius;
// 5 - r;
// 6 - g;
// 7 - b;
// 8 - luminosity;
// 9 - diffusion;
// 10 - specular;

extern "C"
__global__ void render(int width, int height, int len, 
                    int maxSteps, float maxStepDistance,
                    float gravityConstant, float lightSpeed,
                    float cx, float cy, float cz,
                    float rx, float ry, float fov, float specularity,
                    float* data, int8_t* image) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < width && y < height) {
        int index = x + y * width;
        image[index * 3] = -128;
        image[index * 3 + 1] = -128;
        image[index * 3 + 2] = -128;

        float factor = 0.0174532925 * fov;
        float ax = (2.0 * x / width - 1) * factor;
        float ay = (2.0 * y / height - 1) * factor * height / width;

        float cosComp = cos(ry + ay);
        float nx = cos(rx + ax) * cosComp;
        float ny = sin(rx + ax) * cosComp;
        float nz = sin(ry + ay);

        float rpx = cx, rpy = cy, rpz = cz;

        bool work = true;
        int iterations = 0;
        while (iterations < maxSteps && work) {
            float ax = 0, ay = 0, az = 0;
            float minDst = maxStepDistance;

            int i = 0;
            while (i < len && work) {
                float dx = data[i * 11] - rpx;
                float dy = data[i * 11 + 1] - rpy;
                float dz = data[i * 11 + 2] - rpz;
                float dst = pow(dx, 2) + pow(dy, 2) + pow(dz, 2);
                float a = gravityConstant * data[i * 5 + 3] / dst;
                dst = sqrt(dst);
                a /= dst;
                ax += a * dx;
                ay += a * dy;
                az += a * dz;

                dst -= data[i * 11 + 4];
                if (abs(dst) < 1e-1) {
                    if (data[i * 11 + 5] != 0 || data[i * 11 + 6] != 0 || data[i * 11 + 7] != 0) {

                        float spd = gravityConstant * data[i * 11 + 3] / data[i * 11 + 4];
                        if (spd < lightSpeed) {

                            float norx = rpx - data[i * 11];
                            float nory = rpy - data[i * 11 + 1];
                            float norz = rpz - data[i * 11 + 2];
                            float norsqr = norx * norx + nory * nory + norz * norz;

                            float diffR = 0;
                            float diffG = 0;
                            float diffB = 0;

                            float specR = 0;
                            float specG = 0;
                            float specB = 0;

                            float lum = 0;

                            for (int j = 0; j < len; j++) {
                                if (gravityConstant * data[j * 8 + 3] / data[j * 8 + 4] < lightSpeed) {
                                    float lx = data[j * 11] - rpx;
                                    float ly = data[j * 11 + 1] - rpy;
                                    float lz = data[j * 11 + 2] - rpz;
                                    float lsqr = lx * lx + ly * ly + lz * lz;
                                    float diff = max(0.0, (lx * norx + ly * nory + lz * norz) / sqrt(lsqr * norsqr) * data[j * 11 + 9]);

                                    diffR += diff * data[j * 11 + 5];
                                    diffG += diff * data[j * 11 + 6];
                                    diffB += diff * data[j * 11 + 7];

                                    float coef = (lx * norx + ly * nory + lz * norz) / norsqr;
                                    float refx = lx - 2 * norx * coef;
                                    float refy = ly - 2 * nory * coef;
                                    float refz = lz - 2 * norz * coef;

                                    float spec = max(0.0, -(nx * refx + ny * refy + nz * refz) / sqrt(lsqr) * data[j * 11 + 10]);

                                    specR += spec * data[j * 11 + 5];
                                    specG += spec * data[j * 11 + 6];
                                    specB += spec * data[j * 11 + 7];

                                    lum += data[j * 11 + 8];
                                }
                            }
//                            lum /= 10;

                            diffR *= data[i * 11 + 9] / lum;
                            diffG *= data[i * 11 + 9] / lum;
                            diffB *= data[i * 11 + 9] / lum;

                            specR = pow(specR * data[i * 11 + 10] / lum, specularity);
                            specG = pow(specG * data[i * 11 + 10] / lum, specularity);
                            specB = pow(specB * data[i * 11 + 10] / lum, specularity);

                            image[index * 3] = 255.0 * max(0.0, min(1.0, (data[i * 11 + 5] * (data[i * 11 + 8] + diffR) + specR))) - 128;
                            image[index * 3 + 1] = 255.0 * max(0.0, min(1.0, (data[i * 11 + 6] * (data[i * 11 + 8] + diffG) + specG))) - 128;
                            image[index * 3 + 2] = 255.0 * max(0.0, min(1.0, (data[i * 11 + 7] * (data[i * 11 + 8] + diffB) + specB))) - 128;
                        }
                    }
                    work = false;
                }
                else if (dst < minDst) minDst = dst;
                i++;
            }

            if (work) {
                nx += ax / lightSpeed;
                ny += ay / lightSpeed;
                nz += az / lightSpeed;
                float nor = sqrt(nx * nx + ny * ny + nz * nz);
                nx /= nor;
                ny /= nor;
                nz /= nor;

                rpx += nx * minDst;
                rpy += ny * minDst;
                rpz += nz * minDst;
                iterations++;
            }
        }
    }
}